#include "hip/hip_runtime.h"
//===================================================================================================================================
// �C���N���[�h
//===================================================================================================================================
#include "cudaCalc.cuh"
#include "ImguiManager.h"

// Maximum number of threads per block:            1024
// Max dimension size of a thread block(x, y, z) : (1024, 1024, 64)
// Max dimension size of a grid size(x, y, z) : (2147483647, 65535, 65535)
__global__ void CompressWave(float *fData, short *sData);
__device__ void Compress(float *fData, short *sData);

__device__ void Compress(float *fData, short *sData)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	fData[idx] = (float)sData[idx];
}

__global__ void CompressWave(float *fData, short *sData)
{
	Compress(fData, sData);
}

void CUDA_CALC::Kernel1(short *_data, long _size)
{
	//�X���b�h�̐ݒ�
	int blocksizeX = 1024;
	int dataNum = _size / sizeof(short);
	int gridSizeX = dataNum / 1024;
	if (gridSizeX < 2147483647)
	{
		// �v���Z�X
		dim3 grid(gridSizeX, 1, 1);
		dim3 block(blocksizeX, 1, 1);

		// �J�[�l���֐��̌Ăяo��
		int startTime = timeGetTime();

		// �f�o�C�X�������m��(GPU)
		float *fData = nullptr;
		hipError_t hr = hipMalloc((void **)&fData, dataNum * sizeof(float));
		hr = hipMemset(fData, 0, dataNum * sizeof(float));

		short *sData = nullptr;
		hr = hipMalloc((void **)&sData, _size);
		hr = hipMemset(sData, 0, _size);

		float *cData = (float *)malloc(dataNum * sizeof(float));
		memset(cData, 0, dataNum * sizeof(float));

		// �z�X�g->�f�o�C�X
		hr = hipMemcpy(sData, &_data[0], _size, hipMemcpyHostToDevice);

		CompressWave << <grid, block >> > (fData, sData);

		hr = hipMemcpy(cData, &fData[0], dataNum * sizeof(float), hipMemcpyDeviceToHost);

		int usedTime = timeGetTime() - startTime;

		// ��Еt��
		free(cData);
		hipFree(fData);
		hipFree(sData);
	}
}

void CUDA_CALC::tmpPlot(void)
{
	ImVec2 plotextent(ImGui::GetContentRegionAvailWidth(), 100);
	ImGui::PlotLines("", tmpPlotData, imGuiPlotManagerNS::compressSize, 0, "", FLT_MAX, FLT_MAX, plotextent);
}