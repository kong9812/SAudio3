#include "hip/hip_runtime.h"
//===================================================================================================================================
// �C���N���[�h
//===================================================================================================================================
#include "cudaCalc.cuh"
#include "ImguiManager.h"

// Maximum number of threads per block:            1024
// Max dimension size of a thread block(x, y, z) : (1024, 1024, 64)
// Max dimension size of a grid size(x, y, z) : (2147483647, 65535, 65535)
__global__ void CompressWave(float *fData, short *sData, int compressBlock);
__device__ void Compress(float *fData, short *sData, int compressBlock);
__global__ void CompressWave(float *fData, short *sData, int compressBlock)
{
	Compress(fData, sData, compressBlock);
}
__device__ void Compress(float *fData, short *sData, int compressBlock)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	long tmpData = 0;
	for (int i = 0; i < compressBlock; i++)
	{
		tmpData += sData[i + (idx * compressBlock)];
	}
	fData[idx] = (float)tmpData / compressBlock;
}


void CUDA_CALC::Kernel1(short *_data, long _size)
{
	//�X���b�h�̐ݒ�
	int dataNum = 10240;
	int blocksizeX = 1024;
	int gridSizeX = 10240 / 1024;
	if (gridSizeX < 2147483647)
	{
		// �v���Z�X
		dim3 grid(gridSizeX, 1, 1);
		dim3 block(blocksizeX, 1, 1);

		// ���k��
		int compressBlock = ((_size / sizeof(short)) / 10240);

		// �f�o�C�X�������m��(GPU)
		float *fData = nullptr;
		hipError_t hr = hipMalloc((void **)&fData, dataNum * sizeof(float));
		hr = hipMemset(fData, 0, dataNum * sizeof(float));

		short *sData = nullptr;
		hr = hipMalloc((void **)&sData, _size);
		hr = hipMemset(sData, 0, _size);

		// �z�X�g->�f�o�C�X
		hr = hipMemcpy(sData, &_data[0], _size, hipMemcpyHostToDevice);

		int startTime = timeGetTime();
		CompressWave <<<grid, block>>> (fData, sData, compressBlock);
		usedTime = timeGetTime() - startTime;

		hr = hipMemcpy(tmpPlotData, &fData[0], dataNum * sizeof(float), hipMemcpyDeviceToHost);

		// ��Еt��
		hipFree(fData);
		hipFree(sData);
	}
}

void CUDA_CALC::tmpPlot(void)
{
	ImVec2 plotextent(ImGui::GetContentRegionAvailWidth(), 100);
	ImGui::PlotLines("", tmpPlotData, 10240, 0, "", FLT_MAX, FLT_MAX, plotextent);
	ImGui::Text("CUDA usedTime:%d", usedTime);
}