#include "hip/hip_runtime.h"
//===================================================================================================================================
// �C���N���[�h
//===================================================================================================================================
#include "cudaCalc.cuh"

//===================================================================================================================================
// �v���g�^�C�v�錾
//===================================================================================================================================
__global__ void ConversionWave(float *fData, short *sData, int allChannel, int processChannel);
__device__ void Conversion(float *fData, short *sData, int allChannel, int processChannel);
__global__ void CompressWave(float *fData, short *sData, int compressBlock, int allChannel, int processChannel);
__device__ void Compress(float *fData, short *sData, int compressBlock, int allChannel, int processChannel);
__global__ void NormalizeWave(short *sData, short *inData, long inSize, int allChannel, int oldSampleRate, int newSampleRate);
__device__ void Normalize(short *sData, short *inData, long inSize, int allChannel, int oldSampleRate, int newSampleRate);

//===================================================================================================================================
// [CPU->GPU]�ϊ�����
//===================================================================================================================================
__global__ void ConversionWave(float *fData, short *sData, int allChannel, int processChannel)
{
	// [CPU->GPU]�ϊ�����
	Conversion(fData, sData, allChannel, processChannel);
}

//===================================================================================================================================
// [CPU->GPU]�ϊ�����
//===================================================================================================================================
__device__ void Conversion(float *fData, short *sData, int allChannel, int processChannel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	// MAX:1024
	fData[idx] = (float)sData[(idx*allChannel) + processChannel] / (float)SHRT_MAX;
}

//===================================================================================================================================
// �J�[�l�� �ϊ�����
//===================================================================================================================================
Conversion_Data CUDA_CALC::conversion(short *_data, long _size, int channel)
{
	Conversion_Data tmpConversionData = { NULL };
	tmpConversionData.channel = channel;

	// 1�`�����l��������̃T���v�����O��
	tmpConversionData.sampingPerChannel = _size / sizeof(short) / channel;
	
	// �u���b�N(�X���b�hX,�X���b�hY,�X���b�hZ)
	dim3 block(CUDACalcNS::threadX, 1, 1);
	// �O���b�h(�u���b�NX,�u���b�NY)
	dim3 grid(tmpConversionData.sampingPerChannel / block.x, 1, 1);

	// �f�o�C�X�������m��(GPU)
	float *fData = nullptr;
	hipError_t hr = hipMalloc((void **)&fData, sizeof(float)*tmpConversionData.sampingPerChannel);

	short *sData = nullptr;
	hr = hipMalloc((void **)&sData, _size);
	hr = hipMemset(sData, 0, _size);

	// �z�X�g->�f�o�C�X
	hr = hipMemcpy(sData, &_data[0], _size, hipMemcpyKind::hipMemcpyHostToDevice);

	// �J�[�l��+�f�o�C�X->�z�X�g
	tmpConversionData.startTime = timeGetTime();
	tmpConversionData.data = new float *[tmpConversionData.channel];
	for (int i = 0; i < tmpConversionData.channel; i++)
	{
		// ���������Z�b�g
		hr = hipMemset(fData, NULL, sizeof(float)*tmpConversionData.sampingPerChannel);

		tmpConversionData.data[i] = new float[tmpConversionData.sampingPerChannel];
		memset(tmpConversionData.data[i], NULL, sizeof(float)*tmpConversionData.sampingPerChannel);
		// �J�[�l��
		ConversionWave << <grid, block >> > (fData, sData, channel, i);
		hr = hipMemcpy(tmpConversionData.data[i], &fData[0], sizeof(float)*tmpConversionData.sampingPerChannel, hipMemcpyDeviceToHost);
	}
	tmpConversionData.usedTime = timeGetTime() - tmpConversionData.startTime;

	// ��Еt��
	hr = hipFree(fData);
	hr = hipFree(sData);

	return tmpConversionData;
}

//===================================================================================================================================
// [CPU->GPU]���k����
//===================================================================================================================================
__global__ void CompressWave(float *fData, short *sData, int compressBlock, int allChannel, int processChannel)
{
	// [GPU]���k����
	Compress(fData, sData, compressBlock, allChannel, processChannel);
}

//===================================================================================================================================
// [GPU]���k����
//===================================================================================================================================
__device__ void Compress(float *fData, short *sData, int compressBlock, int allChannel, int processChannel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	// MAX:1024

	float tmpData = 0;
	for (int i = 0; i < compressBlock; i++)
	{
		tmpData += (float)sData[((i + (idx * compressBlock))*allChannel) + processChannel] / (float)SHRT_MAX;
	}
	fData[idx] = tmpData / (float)compressBlock;
}

//===================================================================================================================================
// �J�[�l�� ���k����
//===================================================================================================================================
Compress_Data CUDA_CALC::compressor(short *_data, long _size, int channel)
{
	Compress_Data tmpCompressData = { NULL };
	tmpCompressData.channel = channel;
	tmpCompressData.max = 0;
	tmpCompressData.min = 0;

	// �u���b�N(�X���b�hX,�X���b�hY,�X���b�hZ)
	dim3 block(CUDACalcNS::threadX, 1, 1);
	// �O���b�h(�u���b�NX,�u���b�NY)
	dim3 grid(CUDACalcNS::compressSize / block.x, 1, 1);

	// ���k��
	tmpCompressData.compressBlock = ((_size / tmpCompressData.channel / sizeof(short)) / CUDACalcNS::compressSize);

	// �f�o�C�X�������m��(GPU)
	float *fData = nullptr;
	hipError_t hr = hipMalloc((void **)&fData, sizeof(float)*CUDACalcNS::compressSize);

	short *sData = nullptr;
	hr = hipMalloc((void **)&sData, _size);
	hr = hipMemset(sData, 0, _size);

	// �z�X�g->�f�o�C�X
	hr = hipMemcpy(sData, &_data[0], _size, hipMemcpyKind::hipMemcpyHostToDevice);

	// �J�[�l��+�f�o�C�X->�z�X�g
	tmpCompressData.startTime = timeGetTime();
	tmpCompressData.data = new float *[tmpCompressData.channel];
	for (int i = 0; i < tmpCompressData.channel; i++)
	{
		// ���������Z�b�g
		hr = hipMemset(fData, NULL, sizeof(float)*CUDACalcNS::compressSize);

		tmpCompressData.data[i] = new float[CUDACalcNS::compressSize];
		memset(tmpCompressData.data[i], NULL, sizeof(float)*CUDACalcNS::compressSize);
		// �J�[�l��
		CompressWave <<<grid, block>>> (fData, sData, tmpCompressData.compressBlock, channel, i);
		hr = hipMemcpy(tmpCompressData.data[i], &fData[0], sizeof(float)*CUDACalcNS::compressSize, hipMemcpyDeviceToHost);
	}
	for (int i = 0; i < channel; i++)
	{
		for (int j = 0; j < CUDACalcNS::compressSize; j++)
		{
			if (tmpCompressData.data[i][j] > tmpCompressData.max)
			{
				tmpCompressData.max = tmpCompressData.data[i][j];
			}
			if (tmpCompressData.data[i][j] < tmpCompressData.min)
			{
				tmpCompressData.min = tmpCompressData.data[i][j];
			}
		}
	}
	tmpCompressData.usedTime = timeGetTime() - tmpCompressData.startTime;

	// ��Еt��
	hr = hipFree(fData);
	hr = hipFree(sData);

	return tmpCompressData;
}

//===================================================================================================================================
// [CPU->GPU]���K��
//===================================================================================================================================
__global__ void NormalizeWave(short *outData, short *inData, long inSize, int allChannel, int oldSampleRate, int newSampleRate)
{
	Normalize(outData, inData, inSize, allChannel, oldSampleRate, newSampleRate);
}

//===================================================================================================================================
// [GPU]���K��
//===================================================================================================================================
__device__ void Normalize(short *outData, short *inData, long inSize, int allChannel, int oldSampleRate, int newSampleRate)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	// MAX:�V�����T���v����/�`�����l����

	int readPos = (int)((float)idx * ((float)oldSampleRate / (float)newSampleRate));
	float tmpSample = (float)idx * ((float)oldSampleRate / (float)newSampleRate);	// ���Ԃ̃T���v��
	tmpSample -= (int)tmpSample;

	for (int j = 0; j < allChannel; j++)
	{
		// �Ō�̃f�[�^
		if ((idx == (inSize / (int)sizeof(short) / allChannel) - 1))
		{
			outData[idx * allChannel + j] =
				(short)(inData[readPos*allChannel + j]);
		}
		else
		{
			// ���̃f�[�^�����̃f�[�^���傫���Ȃ�
			float tmp1 = inData[readPos * allChannel + j];			// �O�̃f�[�^
			float tmp2 = inData[(readPos + 1) * allChannel + j];	// ���̃f�[�^
			float tmp3 = ((tmp2 - tmp1)*tmpSample + tmp1);			// �������v�Z����
			outData[idx * allChannel + j] = (short)roundf(tmp3);
		}

		//short tmp0 = inData[readPos*allChannel + j];
		//float tmp1 = ((short)inData[readPos*allChannel + j]
		//	- (float)inData[readPos*allChannel + j]) * tmpSample;
		//short tmp3 = (short)(inData[readPos*allChannel + j] + ((float)(inData[readPos*allChannel + j] - inData[readPos*allChannel + j]) * tmpSample));
		//outData[idx * allChannel + j] =
		//	(short)(inData[readPos*allChannel + j]);
	}
	// CPU
	//for (int i = 0; i < (wav.data.waveSize / (int)sizeof(short) / wav.fmt.fmtChannel); i++)
	//{
	//	int		readPos = (int)((float)i * (oldSample / (float)wav.fmt.fmtSampleRate));
	//	float	tmpPos = (float)i * (oldSample / (float)wav.fmt.fmtSampleRate);
	//	tmpPos -= (int)tmpPos;
	//	for (int j = 0; j < wav.fmt.fmtChannel; j++)
	//	{
	//		// �Ō�̃f�[�^
	//		if ((i == (wav.data.waveSize / (int)sizeof(short) / wav.fmt.fmtChannel) - 1))
	//		{
	//			newBuf[i * wav.fmt.fmtChannel + j] =
	//				(short)(wav.data.waveData[readPos*wav.fmt.fmtChannel + j]);
	//		}
	//		else
	//		{
	//			// ���̃f�[�^�����̃f�[�^���傫���Ȃ�
	//			float tmp1 = wav.data.waveData[readPos * wav.fmt.fmtChannel + j];		// �O�̃f�[�^
	//			float tmp2 = wav.data.waveData[(readPos + 1) * wav.fmt.fmtChannel + j];	// ���̃f�[�^
	//			float tmp3 = ((tmp2 - tmp1)*tmpPos + tmp1); // �������v�Z����
	//			newBuf[i * wav.fmt.fmtChannel + j] = (short)roundf(tmp3);
	//		}
	//	}
	//}
}

//===================================================================================================================================
// ���K��
//===================================================================================================================================
short *CUDA_CALC::normalizer(short *_data, long _size, int channel, int oldSampleRate, int newSampleRate)
{
	// ���̒��� = �g�`�̃T�C�Y / ��b������̎��g�� / �`���l���� / short�^�̃T�C�Y
	float soundLengh = ((float)_size / oldSampleRate / channel / sizeof(short));

	// ���K����̃T�C�Y
	long newSize = (long)(newSampleRate * channel * soundLengh * sizeof(short));

	// �f�o�C�X�������m��(GPU)
	short *outData = nullptr;
	hipError_t hr = hipMalloc((void **)&outData, newSize);
	hr = hipMemset(outData, 0, newSize);

	short *inData = nullptr;
	hr = hipMalloc((void **)&inData, _size);
	hr = hipMemset(inData, 0, _size);
	hr = hipMemcpy(inData, &_data[0], _size, hipMemcpyKind::hipMemcpyHostToDevice);

	// �u���b�N(�X���b�hX,�X���b�hY,�X���b�hZ)
	dim3 block(CUDACalcNS::threadX, 1, 1);
	// �O���b�h(�u���b�NX,�u���b�NY)
	dim3 grid(newSize / channel / sizeof(short) / block.x, 1, 1);

	// [CUDA]���K��
	NormalizeWave <<<grid, block>>> (outData, inData, newSize, channel, oldSampleRate, newSampleRate);

	// ���f�[�^
	short *tmp = new short[newSize / sizeof(short)];
	memset(tmp, NULL, newSize);
	hr = hipMemcpy(tmp, &outData[0], newSize, hipMemcpyKind::hipMemcpyDeviceToHost);

	SAFE_DELETE(tmp)

	return nullptr;
}